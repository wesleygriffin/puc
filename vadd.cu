#include "hip/hip_runtime.h"
#include <cstdlib>

__global__ void sum(int n, float* x, float* y) {
  std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t const stride = blockDim.x * gridDim.x;
  for (std::size_t i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <string>

#include <mpi.h>
#include <unistd.h>

int main(int argc, char* argv[]) {
  int N = 0;

  char c;
  while ((c = getopt(argc, argv, "N:")) != -1) {
    switch(c) {
    case 'N':
      try {
        N = std::stoi(optarg);
      } catch (std::exception const& e) {
        std::fprintf(stderr, "invalid argument '%s': %s\n", optarg, e.what());
        std::exit(EXIT_FAILURE);
      }
      break;
    case '?':
    default:
      std::fprintf(stderr, "usage: %s -N <N>\n", argv[0]);
      std::exit(EXIT_FAILURE);
    }
  }

  if (N <= 0) {
    std::fprintf(stderr, "invalid N argument '%d': must be positive\n", N);
    std::exit(EXIT_FAILURE);
  }

  MPI_Init(nullptr, nullptr);

  int worldRank, worldSize;
  MPI_Comm_rank(MPI_COMM_WORLD, &worldRank);
  MPI_Comm_size(MPI_COMM_WORLD, &worldSize);

  float* x = nullptr;
  float* y = nullptr;
  if (worldRank == 0) {
    try {
      x = new float[N * worldSize];
      y = new float[N * worldSize];
    } catch (std::exception const& e) {
      std::fprintf(stderr, "Unable to allocate 2 %zu float arrays\n", N *
                   worldSize);
      MPI_Finalize();
      std::exit(EXIT_FAILURE);
    }

    for (int i = 0; i < N * worldSize; ++i) x[i] = 1.f;
    for (int i = 0; i < N * worldSize; ++i) y[i] = 2.f;
  }

  hipError_t cudaResult;
  int mpiResult;

  float* sub_x = nullptr;
  cudaResult = hipMallocManaged(&sub_x, N * sizeof(float));
  if (cudaResult != hipSuccess) {
    std::fprintf(stderr, "Unable to allocate CUDA managed memory: %s\n",
                 hipGetErrorString(cudaResult));
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  mpiResult = MPI_Scatter(x,             // sendbuf
                          N,             // sendcount
                          MPI_FLOAT,     // sendtype
                          sub_x,         // recvbuf
                          N,             // recvcount (to _each_ process)
                          MPI_FLOAT,     // recvtype
                          0,             // root
                          MPI_COMM_WORLD // comm
                         );
  if (mpiResult != MPI_SUCCESS) {
    int len = 2048;
    char str[len];
    MPI_Error_string(mpiResult, str, &len);
    std::fprintf(stderr, "Unable to scatter memory: %s\n", str);
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  float* sub_y = nullptr;
  cudaResult = hipMallocManaged(&sub_y, N * sizeof(float));
  if (cudaResult != hipSuccess) {
    std::fprintf(stderr, "Unable to allocate CUDA managed memory: %s\n",
                 hipGetErrorString(cudaResult));
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  mpiResult = MPI_Scatter(y,             // sendbuf
                          N,             // sendcount
                          MPI_FLOAT,     // sendtype
                          sub_y,         // recvbuf
                          N,             // recvcount (to _each_ process)
                          MPI_FLOAT,     // recvtype
                          0,             // root
                          MPI_COMM_WORLD // comm
                         );
  if (mpiResult != MPI_SUCCESS) {
    int len = 2048;
    char str[len];
    MPI_Error_string(mpiResult, str, &len);
    std::fprintf(stderr, "Unable to scatter memory: %s\n", str);
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  sum<<<numBlocks, blockSize>>>(N, sub_x, sub_y);

  cudaResult = hipDeviceSynchronize();
  if (cudaResult != hipSuccess) {
    std::fprintf(stderr, "Asynchronous CUDA error: %s\n",
                 hipGetErrorString(cudaResult));
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  float error = 0.f;
  for (int i = 0; i < N; ++i) {
    error = std::max(error, std::abs(sub_y[i] - 3.f));
  }
  std::printf("rank %d error: %g\n", worldRank, error);

  float* errors = nullptr;
  if (worldRank == 0) {
    errors = new float[worldSize];
  }

  mpiResult = MPI_Gather(&error,        // sendbuf
                         1,             // sendcount
                         MPI_FLOAT,     // sendtype
                         errors,        // recvbuf
                         1,             // recvcount (from _each_ process)
                         MPI_FLOAT,     // recvtype
                         0,             // root
                         MPI_COMM_WORLD // comm
                        );
  if (mpiResult != MPI_SUCCESS) {
    int len = 2048;
    char str[len];
    MPI_Error_string(mpiResult, str, &len);
    std::fprintf(stderr, "Unable to gather memory: %s\n", str);
    MPI_Finalize();
    std::exit(EXIT_FAILURE);
  }

  if (worldRank == 0) {
    float globalError = 0.f;
    for (int i = 0; i < worldSize; ++i) {
      globalError = std::max(globalError, errors[i]);
    }
    std::printf("globalError: %g\n", globalError);
  }

  hipFree(sub_y);
  hipFree(sub_x);

  if (worldRank == 0) {
    delete[] errors;
    delete[] y;
    delete[] x;
  }

  MPI_Finalize();
  return EXIT_SUCCESS;
}

